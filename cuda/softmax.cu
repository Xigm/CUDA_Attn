#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "utils.cu"

/* Atomic operations for floating point numbers 

    Definitions at the beginning of the file, if not the compiler explodes

*/

__device__ inline void atomicDivide(float *address, float val) {
    int *address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __float_as_int(__int_as_float(assumed) / val));
    } while (assumed != old);
}

__device__ inline void atomicSubstract(float *address, float val) {
    int *address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __float_as_int(__int_as_float(assumed) - val));
    } while (assumed != old);
}

__device__ inline void atomicExp(float *address) {
    int *address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __float_as_int(expf(__int_as_float(assumed))));
    } while (assumed != old);
}


__global__ void softmax_kernel(float* input, int n_tokens, int batch_size, int n_heads) {

    extern __shared__ float shared_data[];

    int tid = threadIdx.x;
    int b = blockIdx.x; // current batch index
    int h = blockIdx.y; // current head index
    int row = blockIdx.z; // current row index
    int stride = n_tokens * n_tokens * n_heads;

    float max_val;

    // if ((b * stride + h * n_tokens * n_tokens + row * n_tokens + tid) == 0){
    //     printf("Hello from the kernel");
    // }

    // printf("%lu\n", sizeof(float));

    // Reduce to find global max within block
    // printf("%f from tid %d row %d batch %d and head %d sum %d\n", input[b * stride + h * n_tokens * n_tokens + row * n_tokens + tid], tid, row, b, h, b * stride + h * n_tokens * n_tokens + row * n_tokens + tid);
    shared_data[tid] = input[b * stride + h * n_tokens * n_tokens + row * n_tokens + tid];
    // printf("Value: %f\n", shared_data[tid]);
    __syncthreads();
    int half_block = blockDim.x / 2;
    for (int i = half_block; i > 0; i >>= 1) {
        if (tid < i && shared_data[tid + i] > shared_data[tid]) {
            shared_data[tid] = shared_data[tid + i];
            // printf("Max value: %f\n", shared_data[tid]);
        }
        __syncthreads();
    }

    // Broadcast the max value to all threads
    // if (tid == 0) {
    //     printf("Max value: %f of softmax vec %d\n", shared_data[0], b * stride + h * n_tokens * n_tokens + row * n_tokens + tid);
    // }

    max_val = shared_data[0];

    // printf("%d\n", tid);
    // if (tid == 0) {
    //     printf("Max value: %f\n", max_val);
    // }

    // Step 2: Subtract max and exponentiate
    float sum_exp = 0.0;
    atomicSubstract(&input[b * stride + h * n_tokens * n_tokens + row * n_tokens + tid], max_val);

    // printf("Substracted value: %f\n", input[b * stride + h * n_tokens * n_tokens + row * n_tokens + tid]);

    atomicExp(&input[b * stride + h * n_tokens * n_tokens + row * n_tokens + tid]);

    // printf("Exponentiated value: %f\n", input[b * stride + h * n_tokens * n_tokens + row * n_tokens + tid]);

    // Reduce to find the sum of exps
    shared_data[tid] = input[b * stride + h * n_tokens * n_tokens + row * n_tokens + tid];
    __syncthreads();
    for (int i = half_block; i > 0; i >>= 1) {
        if (tid < i) {
            shared_data[tid] += shared_data[tid + i];
        }
        __syncthreads();
    }

    // Broadcast the sum to all threads
    sum_exp = shared_data[0];

    // printf("Sum of exps: %f\n", sum_exp);

    // Step 3: Divide by sum of exps
    atomicDivide(&input[b * stride + h * n_tokens * n_tokens + row * n_tokens + tid], sum_exp);

    // printf("Softmax value: %f\n", input[b * stride + h * n_tokens * n_tokens + row * n_tokens + tid]);

}

void softmax(float**** input, int num_inputs, int batch_size, int n_heads, float**** output) {
    float *d_input;
    float ** p_d_input = &d_input;

    printf("Loading matrix to GPU\n");

    loadMatrixToGPU_batched_sm(input, p_d_input, num_inputs, num_inputs, batch_size, n_heads);

    // print_from_GPU_sm(d_input, num_inputs, num_inputs, batch_size, n_heads);

    // int threads_per_block = 1024;
    // thing is, if num_inputs is low we are losing performance but if it is big...
    // this is better than the previous implementation. Also it is easier to understand
    dim3 block_dims(num_inputs);

    // if (num_inputs > threads_per_block) { 
    //     dim3 grid_dims(batch_size, n_heads, num_inputs / threads_per_block);
    // }
    // else {
    //     dim3 grid_dims(batch_size, n_heads);
    // }

    dim3 grid_dims(batch_size, n_heads, num_inputs);

    // time the kernel execution
    clock_t start, end;
    double cpu_time_used;
    start = clock();
    softmax_kernel<<<grid_dims, block_dims, num_inputs*sizeof(float)>>>(d_input, num_inputs, batch_size, n_heads);
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time taken by GPU kernel: %f seconds\n", cpu_time_used);
    

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    // print from GPU
    // print_from_GPU_sm(d_input, num_inputs, num_inputs, batch_size, n_heads);

    // copy back to cpu
    float * output_array = (float *) malloc(batch_size * num_inputs * num_inputs * n_heads * sizeof(float));
    hipMemcpy(output_array, d_input, batch_size * num_inputs * num_inputs * n_heads * sizeof(float), hipMemcpyDeviceToHost);
    unflattenMatrix(output_array, num_inputs, num_inputs, batch_size, n_heads, output);


}
#include <hip/hip_runtime.h>

#include <stdio.h>

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

void unflattenMatrix(float* flattenedMatrix, int m, int n, int b, float*** matrix) {
  for (int k = 0; k < b; k++) {
    for (int i = 0; i < m; i++) {
      for (int j = 0; j < n; j++) {
        matrix[k][i][j] = flattenedMatrix[m * n * k + i * n + j];
      }
    }
  }
}

void unflattenMatrix(float* flattenedMatrix, int m, int n, float** matrix) {
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      matrix[i][j] = flattenedMatrix[i * n + j];
    }
  }
}


void flattenMatrix(float*** matrix, int m, int n, int b, float* flattenedMatrix) {

    for (int k = 0; k < b; k++) {
      for (int i = 0; i < m; i++) {
          for (int j = 0; j < n; j++) {
              flattenedMatrix[m * n * k + i * n + j] = matrix[k][i][j];  // Correct the index from i * m + j to i * n + j
              // printf("%f ", flattenedMatrix[m * n * k + i * n + j]);
          }
      }
    }
}

void flattenMatrix(float** matrix, int m, int n, float* flattenedMatrix) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            flattenedMatrix[i * n + j] = matrix[i][j];  // Correct the index from i * m + j to i * n + j
            // printf("%f ", flattenedMatrix[i * n + j]);
        }
    }
}


void loadMatrixToGPU(float** matrix, float** deviceMatrix, int m, int n) {
    // Flatten the matrix
    float* flattenedMatrix = (float *)malloc(m * n * sizeof(float));
    flattenMatrix(matrix, m, n, flattenedMatrix);

    size_t size = n * m * sizeof(float);

    // Allocate memory on the GPU
    hipError_t err =  hipMalloc((void**) deviceMatrix, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device matrix: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Copy the flattened matrix from host to device
    hipMemcpy(*deviceMatrix, flattenedMatrix, size, hipMemcpyHostToDevice);


}

void loadMatrixToGPU_batched(float*** matrix, float** deviceMatrix, int m, int n, int b) {
    // Flatten the matrix
    float* flattenedMatrix = (float *)malloc(m * n * b * sizeof(float));
    flattenMatrix(matrix, m, n, b, flattenedMatrix);

    size_t size = b * n * m * sizeof(float);

    // Allocate memory on the GPU
    hipError_t err =  hipMalloc((void**) deviceMatrix, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device matrix: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Copy the flattened matrix from host to device
    hipMemcpy(*deviceMatrix, flattenedMatrix, size, hipMemcpyHostToDevice);


}

void print_from_GPU(float* deviceMatrix, int m, int n, int b) {

    float *d_array = (float *)malloc(m * n * b * sizeof(float));
    hipMemcpy(d_array, deviceMatrix, m * n * b * sizeof(float), hipMemcpyDeviceToHost);
    for (int k = 0; k < b; k++) {
      printf("Batch %d\n", k);
      for (int i = 0; i < m; i++) {
          for (int j = 0; j < n; j++) {
              printf("%f ", d_array[k * m * n + i * n + j]);
          }
          printf("\n");
      }
    }

    free(d_array);
}
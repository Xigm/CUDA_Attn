#include <hip/hip_runtime.h>

#include <stdio.h>

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

void unflattenMatrix(float* flattenedMatrix, int m, int n, int b, float*** matrix) {
  for (int k = 0; k < b; k++) {
    for (int i = 0; i < m; i++) {
      for (int j = 0; j < n; j++) {
        matrix[k][i][j] = flattenedMatrix[m * n * k + i * n + j];
      }
    }
  }
}

void unflattenMatrix(float* flattenedMatrix, int m, int n, int b, int n_heads, float**** matrix) {
  for (int k = 0; k < b; k++) {
    for (int h = 0; h < n_heads; h++){
      for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
          matrix[k][h][i][j] = flattenedMatrix[m * n * n_heads * k + m * n * h + i * n + j];
        }
      }
    }
  }
}

void unflattenMatrix(float* flattenedMatrix, int m, int n, float** matrix) {
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      matrix[i][j] = flattenedMatrix[i * n + j];
    }
  }
}


void flattenMatrix(float*** matrix, int m, int n, int b, float* flattenedMatrix) {

    for (int k = 0; k < b; k++) {
      for (int i = 0; i < m; i++) {
          for (int j = 0; j < n; j++) {
              flattenedMatrix[m * n * k + i * n + j] = matrix[k][i][j];  // Correct the index from i * m + j to i * n + j
              // printf("%f ", flattenedMatrix[m * n * k + i * n + j]);
          }
      }
    }
}

void flattenMatrix_sm(float**** matrix, int m, int n, int b, int n_heads, float* flattenedMatrix) {

  for (int k = 0; k < b; k++) {
    for (int h = 0; h < n_heads; h++){
      for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            flattenedMatrix[m * n * n_heads * k + m * n * h + i * n + j] = matrix[k][h][i][j];  // Correct the index from i * m + j to i * n + j
            // printf("%f ", flattenedMatrix[m * n * n_heads * k + m * n * h + i * n + j]);
        }
      }
    }
  }
}

void flattenMatrix(float** matrix, int m, int n, float* flattenedMatrix) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            flattenedMatrix[i * n + j] = matrix[i][j];  // Correct the index from i * m + j to i * n + j
            // printf("%f ", flattenedMatrix[i * n + j]);
        }
    }
}


void loadMatrixToGPU(float** matrix, float** deviceMatrix, int m, int n) {
    // Flatten the matrix
    float* flattenedMatrix = (float *)malloc(m * n * sizeof(float));
    flattenMatrix(matrix, m, n, flattenedMatrix);

    size_t size = n * m * sizeof(float);

    // Allocate memory on the GPU
    hipError_t err =  hipMalloc((void**) deviceMatrix, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device matrix: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Copy the flattened matrix from host to device
    hipMemcpy(*deviceMatrix, flattenedMatrix, size, hipMemcpyHostToDevice);


}

void loadMatrixToGPU_batched(float*** matrix, float** deviceMatrix, int m, int n, int b) {
    // Flatten the matrix
    float* flattenedMatrix = (float *)malloc(m * n * b * sizeof(float));
    flattenMatrix(matrix, m, n, b, flattenedMatrix);

    size_t size = b * n * m * sizeof(float);

    // Allocate memory on the GPU
    hipError_t err =  hipMalloc((void**) deviceMatrix, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device matrix: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Copy the flattened matrix from host to device
    hipMemcpy(*deviceMatrix, flattenedMatrix, size, hipMemcpyHostToDevice);

}

void loadMatrixToGPU_batched_sm(float**** matrix, float** deviceMatrix, int m, int n, int b, int n_heads) {
    // Flatten the matrix
    float* flattenedMatrix = (float *)malloc(m * n * b * n_heads * sizeof(float));
    flattenMatrix_sm(matrix, m, n, b, n_heads, flattenedMatrix);

    size_t size = b * n_heads * n * m * sizeof(float);

    // Allocate memory on the GPU
    hipError_t err =  hipMalloc((void**) deviceMatrix, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device matrix: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Copy the flattened matrix from host to device
    hipMemcpy(*deviceMatrix, flattenedMatrix, size, hipMemcpyHostToDevice);
    
}

void print_from_GPU(float* deviceMatrix, int m, int n, int b) {

    float *d_array = (float *)malloc(m * n * b * sizeof(float));
    hipMemcpy(d_array, deviceMatrix, m * n * b * sizeof(float), hipMemcpyDeviceToHost);
    for (int k = 0; k < b; k++) {
      printf("Batch %d\n", k);
      for (int i = 0; i < m; i++) {
          for (int j = 0; j < n; j++) {
              printf("%f ", d_array[k * m * n + i * n + j]);
          }
          printf("\n");
      }
    }

    free(d_array);
}

void print_from_GPU_sm(float* deviceMatrix, int m, int n, int b, int n_heads) {

    float *d_array = (float *)malloc(m * n * b * n_heads * sizeof(float));
    hipMemcpy(d_array, deviceMatrix, m * n * b * n_heads * sizeof(float), hipMemcpyDeviceToHost);
    for (int k = 0; k < b; k++) {
      for (int h = 0; h < n_heads; h++){
        printf("Batch %d, Head %d\n", k, h);
        for (int i = 0; i < m; i++) {
            for (int j = 0; j < n; j++) {
                printf("%f ", d_array[k * m * n * n_heads + h * m * n + i * n + j]);
            }
            printf("\n");
        }
      }
     
    }

    free(d_array);
}

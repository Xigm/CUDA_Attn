#include <hip/hip_runtime.h>

#include <stdio.h>

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

void flattenMatrix(float** matrix, int m, int n, float* flattenedMatrix) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            flattenedMatrix[i * n + j] = matrix[i][j];  // Correct the index from i * m + j to i * n + j
            // printf("%f ", flattenedMatrix[i * n + j]);
        }
    }
}


void loadMatrixToGPU(float** matrix, float* deviceMatrix, int m, int n) {
    // Flatten the matrix
    float* flattenedMatrix = (float *)malloc(m * n * sizeof(float));
    flattenMatrix(matrix, m, n, flattenedMatrix);

    size_t size = n * m * sizeof(float);

    // Allocate memory on the GPU
    hipError_t err =  hipMalloc((void**)&deviceMatrix, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device matrix: %s\n", hipGetErrorString(err));
        exit(1);
    }

    // Copy the flattened matrix from host to device
    hipMemcpy(deviceMatrix, flattenedMatrix, size, hipMemcpyHostToDevice);


}
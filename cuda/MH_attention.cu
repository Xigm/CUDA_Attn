#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "utils.cu"
#include "softmax.cu"
#define HIP_INF_F            __int_as_float(0x7f800000)


/* Atomic operations for floating point numbers 

    Definitions at the beginning of the file, if not the compiler explodes

*/

__device__ inline void atomicDivide(float *address, float val) {
    int *address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __float_as_int(__int_as_float(assumed) / val));
    } while (assumed != old);
}

__device__ inline void atomicSubstract(float *address, float val) {
    int *address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __float_as_int(__int_as_float(assumed) - val));
    } while (assumed != old);
}

__device__ inline void atomicMaxfloat(float *address, float val) {
    int *address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
}


__device__ inline void atomicExp(float *address) {
    int *address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __float_as_int(expf(__int_as_float(assumed))));
    } while (assumed != old);
}

__device__ inline void atomicSet(float *address, float value) {
    int *address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        value);
    } while (assumed != old);
}

/* Kernel functions 

    matmul_kernel_batched: kernel for matrix multiplication with batch dimension
    matmul_kernel_semibatched: kernel for matrix multiplication with batch dimension and B matrix without batch dimension
    matmul_kernel_semibatched_debug: kernel for matrix multiplication with batch dimension and B matrix without batch dimension, with debug prints
    matmul_kernel_transposed_batched: kernel for matrix multiplication with transposed B matrix and batch dimension
    normalize: kernel to perform divide by sqrt(dk)
    normalize_atomic: kernel to perform divide by sqrt(dk) with atomic operations
    masked_fill: kernel to fill the upper triangular matrix with -inf
    softmax: kernel to perform softmax
    kernel_exp: kernel to perform exponentiation
    kernel_accumulate: kernel to accumulate the sum of exponentials
    kernel_divide: kernel to divide each element by the sum of exponentials
    
*/

__global__ void matmul_kernel_batched(float* A, float* B, float* C, int m, int n, int p, int batch_size) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < m && y < p && z < batch_size) {
        // printf("Enters kernel\n");
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            // I simply add z * m * n to the index to access the correct batch for A,
            // while B is z * n * p	
            sum += A[z * m * n + x * n + k] * B[z * n * p + k * p + y];
        }
        // here I add z * m * p to the index to access the correct batch for C
        C[z * m * p + x * p + y] = sum;

    }
}

__global__ void matmul_kernel_semibatched(float* A, float* B, float* C, int m, int n, int p, int batch_size) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < m && y < p && z < batch_size) {
        // printf("Enters kernel\n");
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            // I simply add z * m * n to the index to access the correct batch for A,
            // while B does not have batch dim
            sum += A[z * m * n + x * n + k] * B[k * p + y];
        }
        // printf("%d sum %f \n", x * p + y, sum);
        C[z * m * p + x * p + y] = sum;

    }
}




// __global__ void matmul_kernel_semibatched_debug(float* A, float* B, float* C, int m, int n, int p, int batch_size) {
//     int y = blockIdx.y * blockDim.y + threadIdx.y;    
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int z = blockIdx.z * blockDim.z + threadIdx.z;

//     if (x < m && y < p && z < batch_size) {
//         // printf("Enters kernel\n");
//         float sum = 0.0f;
//         for (int k = 0; k < n; k++) {
//             // I simply add z * m * n to the index to access the correct batch for A,
//             // while B does not have batch dim
//             // printf("Batch %d, pos %d, A %f, B %f \n", z, x * p + y, A[z * m * n + x * n + k], B[k * p + y]);
//             // print content of A
//             // printf("Batch %d, pos %d, A %f \n", z, z * m * n + x * n + k, A[z * m * n + x * n + k]);
//             sum += A[z * m * n + x * n + k] * B[k * p + y];
//         }
//         // printf("Batch %d, pos %d, sum %f \n", z, x * p + y, sum);
//         C[z * m * p + x * p + y] = sum;

//     }
// }

// kernel matmul for transposed B matrix
__global__ void matmul_kernel_transposed_batched(float* A, float* B, float* C, int m, int n, int p, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (row < m && col < p && z < batch_size) {
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            sum += A[z * m * n + row * n + k] * B[z * p * n + col * n + k];
        }
        C[z * m * p + row * p + col] = sum;
    }
}

// kernel which performs divide by sqrt(dk)
__global__ void normalize(float* A, int m, int n, int dk) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (row < m && col < n ) {
        A[z * m * n + row * n + col] /= sqrt((float) dk);
    }

}

__global__ void normalize_atomic(float* A, int m, int n, int dk, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (row < m && col < n && z < batch_size) {
        atomicDivide(&A[z * m * n + row * n + col], sqrt((float) dk));
    }

}

// kernel to fill the upper triangular matrix with -inf
__global__ void masked_fill(float* A, int m, int n, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (row < m && col < n && z < batch_size) {
        if (row < col) {
            A[z * m * n + row * n + col] = -HIP_INF_F;
        }
    }
}

// kernel to fill the upper triangular matrix with -inf
__global__ void masked_fill_atomic(float* A, int m, int n, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (row < m && col < n && z < batch_size) {
        if (row < col) {
            atomicSet(&A[z * m * n + row * n + col], -HIP_INF_F);
        }
    }
}


__global__ void softmax(float *input, int rows, int cols) {
    int row = blockIdx.x;  // Each block handles one row
    int tid = threadIdx.x;
    int idx = row * cols + tid;

    if (tid >= cols) return;  // Safeguard against excess threads

    // Step 1: Find max for numerical stability
    extern __shared__ float shared[];
    float *max_val = &shared[0];
    float *exp_sum = &shared[1];
    if (tid == 0) {
        *max_val = input[idx];
        *exp_sum = 0.0;
    }
    __syncthreads();

    for (int i = tid; i < cols; i += blockDim.x) {
        atomicMaxfloat(max_val, input[row * cols + i]);
    }
    __syncthreads();

    // Step 2: Compute sum of exponentials
    float sum_exp = 0.0;
    for (int i = tid; i < cols; i += blockDim.x) {
        sum_exp += expf(input[row * cols + i] - *max_val);
    }

    atomicAdd(exp_sum, sum_exp);
    __syncthreads();

    // Step 3: Calculate softmax output
    for (int i = tid; i < cols; i += blockDim.x) {
        input[row * cols + i] = expf(input[row * cols + i] - *max_val) / *exp_sum;
    }
}


__global__ void kernel_exp(float * z, int num_inputs, int batch_size){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int zz = blockIdx.z * blockDim.z + threadIdx.z;
    int idx = zz * num_inputs * num_inputs + row * num_inputs + col;

    if (row < num_inputs && col < num_inputs && zz < batch_size) {
        atomicExp(&z[idx]);
    }
}

__global__ void kernel_accumulate(float * z, float * sum, int num_inputs, int batch_size){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int zz = blockIdx.z * blockDim.z + threadIdx.z;
    int idx = zz * num_inputs * num_inputs + row * num_inputs + col;
    int idx_sum = idx/num_inputs;

    if (row < num_inputs && col < num_inputs && zz < batch_size) {
        atomicAdd(&sum[idx_sum], z[idx]);
    }
}

__global__ void kernel_divide(float * z, float * sum, int num_inputs, int batch_size){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int zz = blockIdx.z * blockDim.z + threadIdx.z;
    int idx = zz * num_inputs * num_inputs + row * num_inputs + col;
    int idx_sum = idx/num_inputs;

    if (row < num_inputs && col < num_inputs && zz < batch_size) {
        atomicDivide(&z[idx], sum[idx_sum]);
    }
}

__global__ void kernel_substract(float * z, float val, int num_inputs, int batch_size) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int zz = blockIdx.z * blockDim.z + threadIdx.z;
    int idx = zz * num_inputs * num_inputs + row * num_inputs + col;

    if (row < num_inputs && col < num_inputs && zz < batch_size) {
        atomicSubstract(&z[idx], val);
    }

}

void softmax_mig(float *input, int num_inputs, int batch_size, dim3 ks_exp_grid, dim3 ks_exp_block) {

    // added for numerical stability, substract 100
    kernel_substract<<<ks_exp_grid, ks_exp_block>>>(input, (float) 1000, num_inputs, batch_size);

    // Print input array
    // print_from_GPU(input, num_inputs, num_inputs, batch_size);
    
    // write values of input into a file called ./data/degub_softmax.txt 
    FILE* file = fopen("./data/debug_softmax.txt", "w");
    if (file != NULL) {
        float* input_host = (float*)malloc(num_inputs * num_inputs * batch_size * sizeof(float));
        hipMemcpy(input_host, input, num_inputs * num_inputs * batch_size * sizeof(float), hipMemcpyDeviceToHost);
        for (int i = 0; i < num_inputs * num_inputs * batch_size; i++) {
            fprintf(file, "%f\n", input_host[i]);
        }
        fclose(file);
        free(input_host);
    } else {
        printf("Failed to open file for writing.\n");
    }

    // exponentiate all elements
    kernel_exp<<<ks_exp_grid, ks_exp_block>>>(input, num_inputs, batch_size);



    // // Print input array
    print_from_GPU(input, num_inputs, num_inputs, batch_size);

    // sum all elements from a row
    float *sum;
    hipMalloc(&sum, batch_size * num_inputs * sizeof(float));
    kernel_accumulate<<<ks_exp_grid, ks_exp_block>>>(input, sum, num_inputs, batch_size);

    // Copy sum vector from device to host
    // print_from_GPU(sum, num_inputs, 1, batch_size);

    // divide each element by the sum
    kernel_divide<<<ks_exp_grid, ks_exp_block>>>(input, sum, num_inputs, batch_size);

    // printf("\n OUTPUT SOFTMAX \n");
    // print_from_GPU(input, num_inputs, num_inputs, batch_size);

}

// void matmul(float* A, float* B, float* C, int m, int n, int p) {
//     // Allocate device memory
//     float* d_A, *d_B, *d_C;
//     hipMalloc((void**)&d_A, m * n * sizeof(float));
//     hipMalloc((void**)&d_B, n * p * sizeof(float));
//     hipMalloc((void**)&d_C, m * p * sizeof(float));

//     // Copy input matrices from host to device
//     hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(d_B, B, n * p * sizeof(float), hipMemcpyHostToDevice);

//     // Define grid and block dimensions
//     dim3 blockDim(2, 8);
//     dim3 gridDim((p + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);

//     // Launch the matrix multiplication kernel
//     // matmul_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, n, p);

//     // Copy result matrix from device to host
//     hipMemcpy(C, d_C, m * p * sizeof(float), hipMemcpyDeviceToHost);

//     // Free device memory
//     hipFree(d_A);
//     hipFree(d_B);
//     hipFree(d_C);
// }

void attention(float*** input, int num_inputs, int dk, int batch_size, float*** output, float** Wq, float** Wk, float** Wv, float **W_cproj) {
    // allocate device memory for inputs, outputs, weights, and intermediate results attn
    float* d_input, *d_output, *d_output_2, *d_Wq, *d_Wk, *d_Wv, *d_W_cproj, *attn, *Q, *K, *V;
    float ** p_d_input = &d_input;
    float ** p_d_Wq = &d_Wq;
    float ** p_d_Wk = &d_Wk;
    float ** p_d_Wv = &d_Wv;
    float ** p_d_W_cproj = &d_W_cproj;


    loadMatrixToGPU_batched(input, p_d_input, num_inputs, dk, batch_size);
    loadMatrixToGPU(Wq, p_d_Wq, dk, dk);
    loadMatrixToGPU(Wk, p_d_Wk, dk, dk);
    loadMatrixToGPU(Wv, p_d_Wv, dk, dk);
    loadMatrixToGPU(W_cproj, p_d_W_cproj, dk, dk);
    
    hipMalloc(&attn, batch_size * num_inputs * num_inputs * sizeof(float));
    hipMalloc(&Q, batch_size * num_inputs * dk * sizeof(float));
    hipMalloc(&K, batch_size * num_inputs * dk * sizeof(float));
    hipMalloc(&V, batch_size * num_inputs * dk * sizeof(float));
    hipMalloc(&d_output, batch_size * num_inputs * dk * sizeof(float));
    hipMalloc(&d_output_2, batch_size * num_inputs * dk * sizeof(float));

    // if (err6 != hipSuccess) {
    //     fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err6));
    //     exit(EXIT_FAILURE);
    // }
    // define grid and block dimensions
    dim3 blockDim_d_q(dk, num_inputs, batch_size);
    int d1 = 32;
    int d2 = 32;
    int d3 = 32;
    if( num_inputs < d1) {
        d1 = num_inputs;
    }
    if (dk < d2) {
        d2 = dk;
    }
    if (batch_size < d3) {
        d3 = 1;
    }

    dim3 blockDim_d_q_mod(d1, d2, 1);

    int d1g = 1;
    int d2g = 1;
    int d3g = 1;
    if (d1 == 32) {
        d1g = ((int) num_inputs/(32 + 1)) + 1;
    }
    if (d2 == 32) {
        d2g = ((int) dk/(32 + 1)) + 1;
    }
    if (d3 == 32) {
        // d3g = ((int) batch_size/(32 + 1)) + 1;
        d3g = batch_size;
    }

    d3g = batch_size;
    dim3 gridDim(d1g, d2g, d3g);

    // print blockdim and grid dim
    printf("blockDim for matmuls: (%d, %d, %d)\n", blockDim_d_q_mod.x, blockDim_d_q_mod.y, blockDim_d_q_mod.z);
    printf("gridDim for matmuls: (%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);

    // launch the matrix multiplication kernel for Wq

    // print using print_from_GPU(float* deviceMatrix, int m, int n, int b)
    // print_from_GPU(d_input, num_inputs, dk, batch_size);

    matmul_kernel_semibatched<<<gridDim, blockDim_d_q_mod>>>(d_input, d_Wq, Q, num_inputs, dk, dk, batch_size);
    matmul_kernel_semibatched<<<gridDim, blockDim_d_q_mod>>>(d_input, d_Wk, K, num_inputs, dk, dk, batch_size);
    matmul_kernel_semibatched<<<gridDim, blockDim_d_q_mod>>>(d_input, d_Wv, V, num_inputs, dk, dk, batch_size);
    CHECK_KERNELCALL();

    // print Q
    // print_from_GPU(Q, num_inputs, dk, batch_size);


    // launch the matrix multiplication kernel Q * K^T
    // dim3 attn_grid_dim(num_inputs,num_inputs);
    int d = 32;
    if( num_inputs < d) {
        d = num_inputs;
    }

    dim3 attn_block_dim(d, d, 1);

    int dg = 1;
    if (d == 32) {
        dg = ((int) num_inputs/(32 + 1)) + 1;
    }

    dim3 attn_gridDim(dg, dg, batch_size);

    dim3 attn_dim(num_inputs, num_inputs, d3g);

    printf("blockDim for attn: (%d, %d, %d)\n", attn_block_dim.x, attn_block_dim.y, attn_block_dim.z);
    printf("gridDim for attn: (%d, %d, %d)\n", attn_gridDim.x, attn_gridDim.y, attn_gridDim.z);

    matmul_kernel_transposed_batched<<<attn_gridDim, attn_block_dim>>>(Q, K, attn, num_inputs, dk, num_inputs, batch_size);
    CHECK_KERNELCALL();

    // print attn matrix
    // print_from_GPU(attn, num_inputs, num_inputs, batch_size);

    // launch the kernel to perform normalization by sqrt(dk)
    normalize_atomic<<<attn_gridDim, attn_block_dim>>>(attn, num_inputs, num_inputs, dk, batch_size);
    CHECK_KERNELCALL();

    // print attn matrix
    // print_from_GPU(attn, num_inputs, num_inputs, batch_size);

    // launch the kernel to fill the triangular upper matrix with -inf
    masked_fill<<<attn_gridDim, attn_block_dim>>>(attn, num_inputs, num_inputs, batch_size);
    CHECK_KERNELCALL();

    // print attn matrix
    // print_from_GPU(attn, num_inputs, num_inputs, batch_size);

    int vecs_per_block = num_inputs;
    if (num_inputs * num_inputs > 1024) {
        vecs_per_block = 1024/num_inputs;
    }

    dim3 block_dims(num_inputs, vecs_per_block);
    dim3 grid_dims(batch_size, n_heads, (num_inputs-1)/vecs_per_block + 1);

    int tokens_power_2_half = next_power_of_2(num_inputs)/2;

    // launch the kernel for softmaxing the attn matrix
    // softmax<<<num_inputs, num_inputs>>>(attn, num_inputs, num_inputs);
    // softmax_mig(attn, num_inputs, batch_size, attn_gridDim, attn_block_dim);
    softmax_kernel<<<grid_dims, block_dims, vecs_per_block*num_inputs*sizeof(float)>>>(input, n_tokens, batch_size, n_heads, vecs_per_block, next_token_2_half);
    CHECK_KERNELCALL();

    // print attn matrix
    // print_from_GPU(attn, num_inputs, num_inputs, batch_size);


    // launch the matrix multiplication kernel for attn * V
    matmul_kernel_batched<<<gridDim, blockDim_d_q_mod>>>(attn, V, d_output, num_inputs, num_inputs, dk, batch_size);
    CHECK_KERNELCALL();

    // print d_output here
    // print_from_GPU(d_output, num_inputs, dk, batch_size);

    // launch the matrix multiplication kernel for W_cproj
    matmul_kernel_semibatched<<<gridDim, blockDim_d_q_mod>>>(d_output, d_W_cproj, d_output_2, num_inputs, dk, dk, batch_size);
    CHECK_KERNELCALL();

    // print d_output
    // print_from_GPU(d_output_2, num_inputs, dk, batch_size);

    // copy the result matrix from device to host
    float * output_array = (float *) malloc(batch_size * num_inputs * dk * sizeof(float));
    hipMemcpy(output_array, d_output_2, batch_size * num_inputs * dk * sizeof(float), hipMemcpyDeviceToHost);
    unflattenMatrix(output_array, num_inputs, dk, batch_size, output);


    // printf("\n\n");
    // //print output_array
    // for (int i = 0; i < batch_size; i++) {
    //     printf("Batch %d\n", i);
    //     for (int j = 0; j < num_inputs; j++) {
    //         for (int k = 0; k < dk; k++) {
    //             printf("%f ", output[i][j][k]);
    //         }
    //         printf("\n");
    //     }
    // }
    
    // free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_output_2);
    hipFree(d_Wq);
    hipFree(d_Wk);
    hipFree(d_Wv);
    hipFree(d_W_cproj);
    hipFree(attn);
    hipFree(Q);
    hipFree(K);
    hipFree(V);

}
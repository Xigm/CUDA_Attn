#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "attention_batched.cu"

#define MAX_DATA 1000 // Define the maximum number of data points

void attention(float*** input, int num_inputs, int dk, float** output, float** Wq, float** Wk, float** Wv, float** W_cproj);
__global__ void matmul_kernel(float* A, float* B, float* C, int m, int n, int p);
__global__ void matmul_kernel_transposed(float* A, float* B, float* C, int m, int n, int p); 
__global__ void normalize(float* A, int m, int n, int dk);
__global__ void masked_fill(float* A, int m, int n);
void matmul(float* A, float* B, float* C, int m, int n, int p);

void readMatrix(FILE *file, float **matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fscanf(file, "%f", &matrix[i][j]);
        }
    }
}

void readMatrix_batched(FILE *file, float ***matrix, int rows, int cols, int b) {
    for (int k = 0; k < b; k++) {
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                fscanf(file, "%f", &matrix[k][i][j]);
            }
        }
    }
}

int main(int argc, char *argv[]) {
    FILE *inputFile, *outputFile;	// File pointers
    int n_inputs, dk, batch_size;
    float ***inputs, **Wq, **Wk, **Wv, **W_cproj, ***outputs;

    if (argc < 5) {
        fprintf(stderr, "Usage: %s <input file> <output file> <n_inputs> <dk>\n", argv[0]);
        return -1;
    }

    inputFile = fopen(argv[1], "r");
    if (inputFile == NULL) {
        perror("Error opening input file");
        return -1;
    }

    n_inputs = atoi(argv[3]);
    dk = atoi(argv[4]);
    batch_size = atoi(argv[5]);

    printf("n_inputs: %d, dk: %d, batch_size: %d\n", n_inputs, dk, batch_size);

    // Dynamically allocate the matrices
    inputs = (float ***)malloc(batch_size * sizeof(float **));
    Wq = (float **)malloc(dk * sizeof(float *));
    Wk = (float **)malloc(dk * sizeof(float *));
    Wv = (float **)malloc(dk * sizeof(float *));
    W_cproj = (float **)malloc(dk * sizeof(float *));
    outputs = (float ***)malloc(batch_size * sizeof(float **));

    for (int i = 0; i < batch_size; i++) {
        inputs[i] = (float **)malloc(n_inputs * sizeof(float *));
        outputs[i] = (float **)malloc(n_inputs * sizeof(float *));
    }

    for (int i = 0; i < batch_size; i++) {
        for (int j = 0; j < n_inputs; j++) {
            inputs[i][j] = (float *)malloc(dk * sizeof(float));
            outputs[i][j] = (float *)malloc(dk * sizeof(float));
        }
    }

    
    for (int i = 0; i < dk; i++) {
        Wq[i] = (float *)malloc(dk * sizeof(float));
        Wk[i] = (float *)malloc(dk * sizeof(float));
        Wv[i] = (float *)malloc(dk * sizeof(float));
        W_cproj[i] = (float *)malloc(dk * sizeof(float));
    }

    // Load the matrices
    printf("Reading input file\n");
    readMatrix_batched(inputFile, inputs, n_inputs, dk, batch_size);
    readMatrix(inputFile, Wq, dk, dk);
    readMatrix(inputFile, Wk, dk, dk);
    readMatrix(inputFile, Wv, dk, dk);
    readMatrix(inputFile, W_cproj, dk, dk);

    fclose(inputFile);

    clock_t start = clock();
    attention(inputs, n_inputs, dk, batch_size, outputs, Wq, Wk, Wv, W_cproj);
    clock_t end = clock();
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("Time taken by C code: %f seconds\n", seconds);

    // Open the output file for writing
    outputFile = fopen(argv[2], "wb");
    if (outputFile == NULL) {
        perror("Error opening output file");
        fclose(inputFile);
        return -1;
    }

    // Process and write the content of inputs in the output file
    int count = 0;
    printf("%d %d\n", n_inputs, dk);
    for (int k = 0; k < batch_size; k++) {
        for (int i = 0; i < n_inputs; i++) {
            for (int j = 0; j < dk; j++) {
                fprintf(outputFile, "%f ", outputs[k][i][j]);
                count++;
                fprintf(outputFile, "\n");
            }
        }        
    }

    // append the time taken to the output file
    fprintf(outputFile, "%f\n", seconds);

    // Close the file
    fclose(outputFile);

    // Free the dynamically allocated memory
    for (int i = 0; i < batch_size; i++) {
        for (int j = 0; j < n_inputs; j++) {
            free(inputs[i][j]);
            free(outputs[i][j]);
        }
    }
    for (int i = 0; i < dk; i++) {
        free(Wq[i]);
        free(Wk[i]);
        free(Wv[i]);
    }
    free(inputs);
    free(Wq);
    free(Wk);
    free(Wv);

    return 0;

}
